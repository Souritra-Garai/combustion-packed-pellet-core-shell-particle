#include "hip/hip_runtime.h"
#include "lu-solver/Tridiagonal_Matrix.cuh"

#include <iostream>
#include <time.h>

#include <hiprand/hiprand_kernel.h>

#define N 5

__device__ TridiagonalMatrix::Matrix *matrix_A;
__device__ double *vector_x;
__device__ double *vector_b;

__global__ void allocateMemory()
{
	matrix_A = new TridiagonalMatrix::Matrix(N);
	vector_x = new double[N];
	vector_b = new double[N];
}

__global__ void deallocateMemory()
{
	delete [] vector_b;
	delete [] vector_x;
	delete matrix_A;
}

__global__ void print()
{
	matrix_A->print();
}

__global__ void initializeCurandState(unsigned int n, double seed, hiprandState *curand_state_ptr)
{
	unsigned int i = threadIdx.x % n;

	hiprand_init(seed, i, 0, curand_state_ptr + i);
}

__global__ void initializeMatrix(hiprandState *curand_state_ptr)
{
	unsigned int i = threadIdx.x;

	if (i == 0)
	{
		matrix_A->setElement(0, 0, hiprand(curand_state_ptr));
		matrix_A->setElement(0, 1, hiprand(curand_state_ptr));

		vector_x[0] = hiprand(curand_state_ptr);
	}

	else if (i < matrix_A->getDim() - 1)
	{
		matrix_A->setElement(i, i-1, hiprand(curand_state_ptr + i));
		matrix_A->setElement(i, i,   hiprand(curand_state_ptr + i));
		matrix_A->setElement(i, i+1, hiprand(curand_state_ptr + i));

		vector_x[i] = hiprand(curand_state_ptr + i);
	}
	
	else if (i == matrix_A->getDim() - 1)
	{
		matrix_A->setElement(i, i-1, hiprand(curand_state_ptr + i));
		matrix_A->setElement(i, i,   hiprand(curand_state_ptr + i));

		vector_x[i] = hiprand(curand_state_ptr + i);
	}
}

int main(int argc, char const *argv[])
{
	hiprandState *curand_states;
	hipMalloc(&curand_states, N * sizeof(hiprandState));

	allocateMemory<<<1,1>>>();

	TridiagonalMatrix::Matrix *matrix_A;
	double *vector_x, *vector_b;

	hipMemcpyFromSymbol(&vector_x, HIP_SYMBOL(::vector_x), sizeof(double *));
	hipMemcpyFromSymbol(&vector_b, HIP_SYMBOL(::vector_b), sizeof(double *));
	hipMemcpyFromSymbol(&matrix_A, HIP_SYMBOL(::matrix_A), sizeof(TridiagonalMatrix::Matrix *));

	initializeCurandState<<<1,N>>>(N, time(0), curand_states);

	hipDeviceSynchronize();

	initializeMatrix<<<1,N>>>(curand_states);

	hipDeviceSynchronize();

	TridiagonalMatrix::multiply<<<1,N>>>(matrix_A, vector_x, vector_b);

	hipDeviceSynchronize();

	print<<<1,1>>>();

	deallocateMemory<<<1,1>>>();

	hipFree(curand_states);

	return 0;
}

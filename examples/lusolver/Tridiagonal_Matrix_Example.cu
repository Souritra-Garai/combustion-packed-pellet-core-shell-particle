#include "hip/hip_runtime.h"
#include "lusolver/Tridiagonal_Matrix.hpp"

#include <iostream>
#include <time.h>

#include <hiprand/hiprand_kernel.h>

__global__ void initializeCurandState(unsigned int n, double seed, hiprandState *curand_state_ptr)
{
	unsigned int i = threadIdx.x % n;

	hiprand_init(seed, i, 0, curand_state_ptr + i);
}

__global__ void initializeMatrix(unsigned int n, double *matrix_ptr, double *vector_ptr, hiprandState *curand_state_ptr)
{
	unsigned int i = threadIdx.x;

	if (i == 0)
	{
		matrix_ptr[getTridiagonalMatrixIndex(0, 0)] = hiprand(curand_state_ptr);
		matrix_ptr[getTridiagonalMatrixIndex(0, 1)] = hiprand(curand_state_ptr);

		vector_ptr[0] = hiprand(curand_state_ptr);
	}

	else if (i < n-1)
	{
		matrix_ptr[getTridiagonalMatrixIndex(i, i-1)] = hiprand(curand_state_ptr + i);
		matrix_ptr[getTridiagonalMatrixIndex(i, i)]   = hiprand(curand_state_ptr + i);
		matrix_ptr[getTridiagonalMatrixIndex(i, i+1)] = hiprand(curand_state_ptr + i);

		vector_ptr[i] = hiprand(curand_state_ptr + i);
	}
	
	else if (i == n-1)
	{
		matrix_ptr[getTridiagonalMatrixIndex(n-1, n-2)] = hiprand(curand_state_ptr + n-1);
		matrix_ptr[getTridiagonalMatrixIndex(n-1, n-1)] = hiprand(curand_state_ptr + n-1);

		vector_ptr[n-1] = hiprand(curand_state_ptr + n-1);
	}
}

int main(int argc, char const *argv[])
{
	const unsigned int n = 5;

	double *tridiagonal_matrix, *x, *b;

	hiprandState *curand_states;

	hipMalloc(&x, n * sizeof(double));
	hipMalloc(&b, n * sizeof(double));
	hipMalloc(&tridiagonal_matrix, getTridiagonalMatrixSize(n) * sizeof(double));

	hipMalloc(&curand_states, n * sizeof(hiprandState));

	initializeCurandState<<<1,n>>>(n, time(0), curand_states);

	hipDeviceSynchronize();

	initializeMatrix<<<1,n>>>(n, tridiagonal_matrix, x, curand_states);

	hipDeviceSynchronize();

	multiplyTridiagonalMatrix<<<1,n>>>(n, tridiagonal_matrix, x, b);

	printTridiagonalMatrix(std::cout, n, tridiagonal_matrix);

	hipFree(x);
	hipFree(b);
	hipFree(tridiagonal_matrix);
	hipFree(curand_states);

	return 0;
}

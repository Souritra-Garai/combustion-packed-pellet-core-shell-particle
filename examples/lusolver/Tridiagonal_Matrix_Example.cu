#include "hip/hip_runtime.h"
#include "lusolver/Tridiagonal_Matrix.cuh"

#include <iostream>
#include <time.h>

#include <hiprand/hiprand_kernel.h>

__global__ void initializeCurandState(unsigned int n, double seed, hiprandState *curand_state_ptr)
{
	unsigned int i = threadIdx.x % n;

	hiprand_init(seed, i, 0, curand_state_ptr + i);
}

__global__ void initializeMatrix(TridiagonalMatrix::Matrix A, double *vector_ptr, hiprandState *curand_state_ptr)
{
	unsigned int i = threadIdx.x;

	if (i == 0)
	{
		A.setElement(0, 0, hiprand(curand_state_ptr));
		A.setElement(0, 1, hiprand(curand_state_ptr));

		vector_ptr[0] = hiprand(curand_state_ptr);
	}

	else if (i < A.getDim() - 1)
	{
		A.setElement(i, i-1, hiprand(curand_state_ptr + i));
		A.setElement(i, i,   hiprand(curand_state_ptr + i));
		A.setElement(i, i+1, hiprand(curand_state_ptr + i));

		vector_ptr[i] = hiprand(curand_state_ptr + i);
	}
	
	else if (i == A.getDim() - 1)
	{
		A.setElement(i, i-1, hiprand(curand_state_ptr + i));
		A.setElement(i, i,   hiprand(curand_state_ptr + i));

		vector_ptr[i] = hiprand(curand_state_ptr + i);
	}
}

int main(int argc, char const *argv[])
{
	const unsigned int n = 5;

	double *x, *b;

	hiprandState *curand_states;

	hipMalloc(&x, n * sizeof(double));
	hipMalloc(&b, n * sizeof(double));

	hipMalloc(&curand_states, n * sizeof(hiprandState));

	TridiagonalMatrix::Matrix matrix;
	matrix.allocateMemoryFromHost(n);

	initializeCurandState<<<1,n>>>(n, time(0), curand_states);

	hipDeviceSynchronize();

	initializeMatrix<<<1,n>>>(matrix, x, curand_states);

	hipDeviceSynchronize();

	TridiagonalMatrix::multiply<<<1,n>>>(matrix, x, b);

	hipDeviceSynchronize();

	matrix.print(std::cout);

	hipFree(x);
	hipFree(b);
	hipFree(curand_states);

	matrix.deallocateMemoryFromHost();

	return 0;
}

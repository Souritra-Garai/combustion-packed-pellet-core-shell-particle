#include "hip/hip_runtime.h"
#include "lusolver/LU_Solver.cuh"

#include <iostream>
#include <time.h>

#include <hiprand/hiprand_kernel.h>

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* a, double b) { return b; }
#endif

__global__ void initializeCurandState(unsigned int n, double seed, hiprandState *curand_state_ptr)
{
	unsigned int i = threadIdx.x % n;

	hiprand_init(seed, i, 0, curand_state_ptr + i);
}

__global__ void initializeMatrix(TridiagonalMatrix::Matrix A, double *vector_ptr, hiprandState *curand_state_ptr)
{
	unsigned int i = threadIdx.x;

	if (i == 0)
	{
		A.setElement(0, 0, hiprand(curand_state_ptr));
		A.setElement(0, 1, hiprand(curand_state_ptr));

		vector_ptr[0] = hiprand(curand_state_ptr);
	}

	else if (i < A.getDim() - 1)
	{
		A.setElement(i, i-1, hiprand(curand_state_ptr + i));
		A.setElement(i, i,   hiprand(curand_state_ptr + i));
		A.setElement(i, i+1, hiprand(curand_state_ptr + i));

		vector_ptr[i] = hiprand(curand_state_ptr + i);
	}
	
	else if (i == A.getDim() - 1)
	{
		A.setElement(i, i-1, hiprand(curand_state_ptr + i));
		A.setElement(i, i,   hiprand(curand_state_ptr + i));

		vector_ptr[i] = hiprand(curand_state_ptr + i);
	}
}

__global__ void initializeSolver(TridiagonalMatrix::Matrix A, double *b, LUSolver solver)
{
	unsigned int i = threadIdx.x;

	if (i==0)

		solver.setEquationFirstRow(A.getElement(0, 0), A.getElement(0, 1), b[0]);

	else if (i < solver.getDim() - 1)

		solver.setEquation(i, A.getElement(i, i-1), A.getElement(i, i), A.getElement(i, i+1), b[i]);

	else if (i == solver.getDim() - 1)

		solver.setEquationLastRow(A.getElement(i, i-1), A.getElement(i, i), b[i]);
}

__global__ void solve(LUSolver solver, double *x)
{
	solver.getSolution(x);
}

__global__ void calcError(unsigned int n, double *x, double *y, double *sum)
{
	unsigned int i = threadIdx.x;
	
	double error = (x[i] - y[i]) * (x[i] - y[i]);

	// printf("%f\n", error);

	atomicAdd(sum, error);
}

int main(int argc, char const *argv[])
{
	const unsigned int n = 1024;

	double *x, *b;

	hiprandState *curand_states;

	hipMalloc(&x, n * sizeof(double));
	hipMalloc(&b, n * sizeof(double));

	hipMalloc(&curand_states, n * sizeof(hiprandState));

	TridiagonalMatrix::Matrix matrix;
	matrix.allocateMemoryFromHost(n);

	initializeCurandState<<<1,n>>>(n, time(0), curand_states);

	hipDeviceSynchronize();

	initializeMatrix<<<1,n>>>(matrix, x, curand_states);

	hipDeviceSynchronize();

	TridiagonalMatrix::multiply<<<1,n>>>(matrix, x, b);

	hipDeviceSynchronize();

    LUSolver my_solver;
	my_solver.allocateMemoryFromHost(n);

    initializeSolver<<<1,n>>>(matrix, b, my_solver);

	hipDeviceSynchronize();

	// my_solver.printMatrixEquation(std::cout);

    double *x_soln;

	hipMalloc(&x_soln, n * sizeof(double));

    solve<<<1,1>>>(my_solver, x_soln);

	hipDeviceSynchronize();

    double *MSE, *MSE_h;

	hipMalloc(&MSE, sizeof(double));

	MSE_h = new double;
	*MSE_h = 0;

	hipMemcpy(MSE, MSE_h, sizeof(double), hipMemcpyHostToDevice);

	calcError<<<1,n>>>(n, x, x_soln, MSE);

	hipDeviceSynchronize();

	hipMemcpy(MSE_h, MSE, sizeof(double), hipMemcpyDeviceToHost);

    std::cout << "\nMSE : " << *MSE_h << std::endl;

	hipFree(x_soln);
	hipFree(MSE);
	delete MSE_h;

	hipFree(x);
	hipFree(b);
	hipFree(curand_states);

	matrix.deallocateMemoryFromHost();
	my_solver.deallocateMemoryFromHost();

    return 0;
}
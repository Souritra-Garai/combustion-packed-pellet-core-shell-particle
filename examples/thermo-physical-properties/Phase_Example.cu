#include "hip/hip_runtime.h"
#include "thermo-physical-properties/Phase.cuh"

#include <iostream>

#define N 1000
#define TEMPERATURE_LOWER_BOUND	250.0	// K
#define TEMPERATURE_UPPER_BOUND 1000.0	// K
#define GET_TEMPERATURE(i) (TEMPERATURE_LOWER_BOUND + (TEMPERATURE_UPPER_BOUND - TEMPERATURE_LOWER_BOUND) * ((double) i) / ((double) N - 1.0))

__device__ Phase *species_phase;

__global__ void allocateMemory()
{
	Enthalpy species_enthalpy(
		28.08920,
		-5.414849,
		8.560423,
		3.427370,
		-0.277375,
		-9.147187
	);

	ThermalConductivity species_thermal_conductivity(248.0, -0.067, 0.0);

	species_phase = new Phase(
		2700.0,
		species_enthalpy,
		species_thermal_conductivity,
		273.15,
		933.0,
		10
	);
}

__global__ void deallocateMemory()
{
	delete species_phase;
}

__global__ void calcEnthalpies(double *enthalpy_array, double *heat_capacity_array, double *thermal_conductivity_array)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N) 
	{
		double T = GET_TEMPERATURE(i);
		enthalpy_array[i] = species_phase->getEnthalpy(T);
		heat_capacity_array[i] = species_phase->getHeatCapacity(T);
		thermal_conductivity_array[i] = species_phase->getThermalConductivity(T);
	}
}

int main(int argc, char const *argv[])
{
	double enthalpy_array_h[N], heat_capacity_array_h[N], thermal_conductivity_array_h[N];

	double *enthalpy_array_d, *heat_capacity_array_d, *thermal_conductivity_array_d;
	hipMalloc(&enthalpy_array_d, N*sizeof(double));
	hipMalloc(&heat_capacity_array_d, N*sizeof(double));
	hipMalloc(&thermal_conductivity_array_d, N*sizeof(double));

	allocateMemory<<<1,1>>>();

	hipDeviceSynchronize();

	calcEnthalpies<<<(N+255)/256, 256>>>(enthalpy_array_d, heat_capacity_array_d, thermal_conductivity_array_d);

	hipDeviceSynchronize();
	
	hipMemcpy(enthalpy_array_h, enthalpy_array_d, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(heat_capacity_array_h, heat_capacity_array_d, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(thermal_conductivity_array_h, thermal_conductivity_array_d, N * sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)

		std::cout << "Temperature : " << GET_TEMPERATURE(i) << " K\t" << "Heat Capacity : " << heat_capacity_array_h[i] << " J / mol. - K\t" <<
		"Enthalpy : " << enthalpy_array_h[i] << " J / mol.\t" << "Thermal Conductivity : " << thermal_conductivity_array_h[i] << " W / m - K\n";

	deallocateMemory<<<1,1>>>();

	hipFree(thermal_conductivity_array_d);
	hipFree(heat_capacity_array_d);
	hipFree(enthalpy_array_d);

	return 0;
}

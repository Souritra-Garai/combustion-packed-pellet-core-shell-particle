#include "hip/hip_runtime.h"
#include "thermo-physical-properties/Ideal_Gas.cuh"

#include <iostream>

#define N 1000
#define TEMPERATURE_LOWER_BOUND	250.0	// K
#define TEMPERATURE_UPPER_BOUND 1000.0	// K
#define GET_TEMPERATURE(i) (TEMPERATURE_LOWER_BOUND + (TEMPERATURE_UPPER_BOUND - TEMPERATURE_LOWER_BOUND) * ((double) i) / ((double) N - 1.0))

__device__ IdealGas *gas;

__global__ void allocateMemory()
{
	Enthalpy gas_enthalpy(
		20.78600,
		2.825911E-7,
		-1.464191E-7,
		1.092131E-8,
		-3.661371E-8,
		-6.197350
	);

	ThermalConductivity gas_thermal_conductivity(1.49E-3, 5.98E-5, -1.92E-8);

	gas = new IdealGas(
		39.948E-3,
		gas_enthalpy,
		gas_thermal_conductivity
	);
}

__global__ void deallocateMemory()
{
	delete gas;
}

__global__ void calcEnthalpies(double *enthalpy_array, double *heat_capacity_array, double *thermal_conductivity_array)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N) 
	{
		double T = GET_TEMPERATURE(i);
		enthalpy_array[i] = gas->getEnthalpy(T);
		heat_capacity_array[i] = gas->getHeatCapacity(T);
		thermal_conductivity_array[i] = gas->getThermalConductivity(T);
	}
}

int main(int argc, char const *argv[])
{
	double enthalpy_array_h[N], heat_capacity_array_h[N], thermal_conductivity_array_h[N];

	double *enthalpy_array_d, *heat_capacity_array_d, *thermal_conductivity_array_d;
	hipMalloc(&enthalpy_array_d, N*sizeof(double));
	hipMalloc(&heat_capacity_array_d, N*sizeof(double));
	hipMalloc(&thermal_conductivity_array_d, N*sizeof(double));

	allocateMemory<<<1,1>>>();

	hipDeviceSynchronize();

	calcEnthalpies<<<(N+255)/256, 256>>>(enthalpy_array_d, heat_capacity_array_d, thermal_conductivity_array_d);

	hipDeviceSynchronize();
	
	hipMemcpy(enthalpy_array_h, enthalpy_array_d, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(heat_capacity_array_h, heat_capacity_array_d, N * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(thermal_conductivity_array_h, thermal_conductivity_array_d, N * sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)

		std::cout << "Temperature : " << GET_TEMPERATURE(i) << " K\t" << "Heat Capacity : " << heat_capacity_array_h[i] << " J / kg - K\t" <<
		"Enthalpy : " << enthalpy_array_h[i] << " J / kg\t" << "Thermal Conductivity : " << thermal_conductivity_array_h[i] << " W / m - K\n";

	deallocateMemory<<<1,1>>>();

	hipFree(thermal_conductivity_array_d);
	hipFree(heat_capacity_array_d);
	hipFree(enthalpy_array_d);

	return 0;
}
